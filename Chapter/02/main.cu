#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void helloCUDA(void) {
    printf("Hello CUDA from GPU!\n");
}

int main(void) {
    printf("Hello GPU from CPU!\n");
    helloCUDA<<<1, 4>>>(); // Block: 1; Thread: 4
    hipDeviceSynchronize();
    return 0;
}


